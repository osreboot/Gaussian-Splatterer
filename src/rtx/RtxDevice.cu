#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "RtxDevice.cuh"

using namespace owl;

// Selects a random point inside a unit sphere
inline __device__ vec3f randomUnitSphere(LCG<4>& random) {
    vec3f v;
    do {
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    } while(dot(v, v) >= 1.0f);
    return v;
}

// Tracks a ray through multiple bounces in the world
inline __device__ vec3f tracePath(const RayGenerator& rayGen, Ray& ray, PerRayData& prd) {
    vec3f colorAtten = vec3f(1.0f);

    bool hasReflected = false;

    // Loop as long as we haven't reached the maximum bounce depth
    for (int i = 0; i < 50; i++) {
        prd.shouldTerminate = true;
        prd.reflected = false;

        // Launch the ray
        traceRay(rayGen.worldHandle, ray, prd);

        /*
        vec3f colorAccum3 = vec3f(colorAccum.x, colorAccum.y, colorAccum.z) + vec3f(prd.color.x, prd.color.y, prd.color.z) * (1.0f - colorAccum.w);
        float colorAccumA = 1.0f - (1.0f - colorAccum.w) * (1.0f - prd.color.w);
        colorAccum = vec4f(colorAccum3, colorAccumA);*/

        colorAtten *= vec3f(prd.color.x, prd.color.y, prd.color.z);
        if (prd.reflected) hasReflected = true;

        if (i == 0) {
            for (int c = 0; c < rayGen.splatCamerasCount; c++) {
                if(!prd.shouldTerminate && dot(ray.direction, rayGen.splatCameras[c] - ray.origin) > dot(ray.direction, prd.hitOrigin - ray.origin)) continue;

                vec3f rayClosest = ray.origin + ray.direction * dot(ray.direction, rayGen.splatCameras[c] - ray.origin);
                vec3f rayDelta = rayGen.splatCameras[c] - rayClosest;
                if (dot(rayDelta, rayDelta) < SPLAT_CAMERA_DOT_SIZE * SPLAT_CAMERA_DOT_SIZE) {
                    prd.splatTouchesCamera = true;
                    break;
                }
            }
        }

        // The ray hit the sky or a light source
        if (prd.shouldTerminate) return hasReflected ? colorAtten : rayGen.background;

        // Re-initialize the ray based on collision parameters
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);
    }

    // Max bounces exceeded
    return {0.0f, 0.0f, 0.0f};
}

// Ray generation program
OPTIX_RAYGEN_PROGRAM(rayGenProgram)() {
    const RayGenerator& rayGen = getProgramData<RayGenerator>();
    const vec2i pixel = getLaunchIndex();

    vec3f color = vec3f(0.0f);
    PerRayData prd;

    // Cast rays to fulfill the number of required samples
    for(int i = 0; i < rayGen.samples; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = rayGen.cameraLocation;

        const vec2f pixelFine = vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5);
        const vec3f viewFarZ = vec3f((pixelFine.x * 2.0 / rayGen.size.x) - 1.0f, (pixelFine.y * 2.0 / rayGen.size.y) - 1.0f, 1.0f);
        vec4f rayFarZ = vec4f(viewFarZ.x * rayGen.cameraMatrix[0] + viewFarZ.y * rayGen.cameraMatrix[4] + viewFarZ.z * rayGen.cameraMatrix[8] + rayGen.cameraMatrix[12],
                              viewFarZ.x * rayGen.cameraMatrix[1] + viewFarZ.y * rayGen.cameraMatrix[5] + viewFarZ.z * rayGen.cameraMatrix[9] + rayGen.cameraMatrix[13],
                              viewFarZ.x * rayGen.cameraMatrix[2] + viewFarZ.y * rayGen.cameraMatrix[6] + viewFarZ.z * rayGen.cameraMatrix[10] + rayGen.cameraMatrix[14],
                              viewFarZ.x * rayGen.cameraMatrix[3] + viewFarZ.y * rayGen.cameraMatrix[7] + viewFarZ.z * rayGen.cameraMatrix[11] + rayGen.cameraMatrix[15]);

        ray.direction = normalize(vec3f(rayFarZ.x / rayFarZ.w, rayFarZ.y / rayFarZ.w, rayFarZ.z / rayFarZ.w) - rayGen.cameraLocation);

        // Trace the ray's path
        vec3f colorOut = tracePath(rayGen, ray, prd) * PROGRAM_EXPOSURE_FACTOR;

        // Clamp the output color
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    color /= (float)rayGen.samples;

    if (prd.splatTouchesCamera) color = vec3f(1.0f) - color;

    // Assign frame buffer pixel color based on average of all samples
    rayGen.frameBuffer[pixel.x + rayGen.size.x * pixel.y] = make_rgba(color);
}

// Ray hit program
OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
    PerRayData& prd = getPRD<PerRayData>();
    const WorldGeometry& world = getProgramData<WorldGeometry>();

    // Fetch data about the collision surface
    const unsigned int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = world.triangles[indexPrimitive];
    const Material material = {false, 0.0f, 0.0f, {1.0f, 1.0f, 1.0f}, 0.0f, 0};

    // Calculate the normal of the surface
    const vec3f normalSurface = normalize(cross(world.vertices[index.y] - world.vertices[index.x],
                                                world.vertices[index.z] - world.vertices[index.x]));

    const vec3f ro = optixGetWorldRayOrigin();
    const vec3f rd = optixGetWorldRayDirection();
    const vec3f rdn = normalize(rd);

    vec2f uv = optixGetTriangleBarycentrics();
    vec2f textureCoord = (1.0f - uv.x - uv.y) * world.textureCoords[indexPrimitive * 3] +
                         uv.x * world.textureCoords[indexPrimitive * 3 + 1] +
                         uv.y * world.textureCoords[indexPrimitive * 3 + 2];
    vec4f colorTexture = vec4f(tex2D<float4>(world.textures[TEXTURE_DIFFUSE], textureCoord.x, 1.0f - textureCoord.y));

    // Calculate the position of the collision
    prd.hitOrigin = ro + optixGetRayTmax() * rd;

    if (colorTexture.w > prd.random()) { // Ray collided with the material
        // Calculate reflected direction
        vec3f directionReflect = rd - 2.0f * dot(rd, normalSurface) * normalSurface;
        if(prd.random() > material.reflectivity){ // Scattering for lambertians
            directionReflect = normalSurface + randomUnitSphere(prd.random);
        }

        // Assign final ray data based on all the above calculations
        prd.bounceDirection = directionReflect;

        // Diffuse material scattering
        prd.bounceDirection += material.diffuse * randomUnitSphere(prd.random);

        prd.color = material.color * vec3f(colorTexture.x, colorTexture.y, colorTexture.z);
        prd.reflected = true;
    } else { // Ray passed through the material
        prd.bounceDirection = rdn;
        prd.color = vec3f(1.0f);
    }

    prd.shouldTerminate = material.fullbright;
}

// Ray miss program
OPTIX_MISS_PROGRAM(miss)() {
    PerRayData& prd = getPRD<PerRayData>();

    prd.shouldTerminate = true;

    vec3f rayNormal = normalize(vec3f(optixGetWorldRayDirection()));
    prd.color = vec3f(min(1.0f, 1.0f + rayNormal.y));
    //prd.color = vec3f(0.0f, 0.0f, 0.0f);
}
