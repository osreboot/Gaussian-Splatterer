#include "hip/hip_runtime.h"
#include "RtxDevice.cuh"

#include <optix_device.h>

using namespace owl;

// Selects a random point inside a unit sphere
inline __device__ vec3f randomUnitSphere(LCG<4>& random) {
    vec3f v;
    do {
        v = 2.0f * vec3f(random(), random(), random()) - vec3f(1.0f, 1.0f, 1.0f);
    } while(dot(v, v) >= 1.0f);
    return v;
}

// Tracks a ray through multiple bounces in the world
inline __device__ vec3f tracePath(const RayGenerator& rayGen, Ray& ray, PerRayData& prd) {
    vec3f colorAtten = vec3f(1.0f);

    bool hasReflected = false;

    // Loop as long as we haven't reached the maximum bounce depth
    for (int i = 0; i < 50; i++) {
        prd.shouldTerminate = true;
        prd.reflected = false;

        // Launch the ray
        traceRay(rayGen.worldHandle, ray, prd);

        // Apply ray color attenuation
        colorAtten *= vec3f(prd.color.x, prd.color.y, prd.color.z);
        if (prd.reflected) hasReflected = true;

        // Detect if the ray hit a splat camera indicator orb. This is inefficient and causes some lag, a better
        // solution would be to use proper hardware acceleration via a custom geometry type.
        if (i == 0) { // Only check on the first iteration because we don't want the camera orbs to cast light/shadows.
            for (int c = 0; c < rayGen.splatCamerasCount; c++) {
                if(!prd.shouldTerminate && dot(ray.direction, rayGen.splatCameras[c] - ray.origin) > dot(ray.direction, prd.hitOrigin - ray.origin)) continue;

                vec3f rayClosest = ray.origin + ray.direction * dot(ray.direction, rayGen.splatCameras[c] - ray.origin);
                vec3f rayDelta = rayGen.splatCameras[c] - rayClosest;
                if (dot(rayDelta, rayDelta) < SPLAT_CAMERA_DOT_SIZE * SPLAT_CAMERA_DOT_SIZE) {
                    prd.splatTouchesCamera = true;
                    break;
                }
            }
        }

        // The ray hit the sky or a light source
        if (prd.shouldTerminate) return hasReflected ? colorAtten : rayGen.background;

        // Re-initialize the ray based on collision parameters
        ray = Ray(prd.hitOrigin, prd.bounceDirection, 1e-3f, 1e10f);
    }

    // Max bounces exceeded
    return {0.0f, 0.0f, 0.0f};
}

// Ray generation program
OPTIX_RAYGEN_PROGRAM(rayGenProgram)() {
    const RayGenerator& rayGen = getProgramData<RayGenerator>();
    const vec2i pixel = getLaunchIndex();

    vec3f color = vec3f(0.0f);
    PerRayData prd;

    // Cast rays to fulfill the number of required samples
    for(int i = 0; i < rayGen.samples; i++){
        // Create ray from camera
        Ray ray;
        ray.origin = rayGen.cameraLocation;

        // Calculate the ray's direction based on the camera matrix
        const vec2f pixelFine = vec2f(pixel) + vec2f(prd.random(), prd.random()) + vec2f(0.5);
        const vec3f viewFarZ = vec3f((pixelFine.x * 2.0 / rayGen.size.x) - 1.0f, (pixelFine.y * 2.0 / rayGen.size.y) - 1.0f, 1.0f);
        const vec4f rayFarZ = vec4f(viewFarZ.x * rayGen.cameraMatrix[0] + viewFarZ.y * rayGen.cameraMatrix[4] + viewFarZ.z * rayGen.cameraMatrix[8] + rayGen.cameraMatrix[12],
                                    viewFarZ.x * rayGen.cameraMatrix[1] + viewFarZ.y * rayGen.cameraMatrix[5] + viewFarZ.z * rayGen.cameraMatrix[9] + rayGen.cameraMatrix[13],
                                    viewFarZ.x * rayGen.cameraMatrix[2] + viewFarZ.y * rayGen.cameraMatrix[6] + viewFarZ.z * rayGen.cameraMatrix[10] + rayGen.cameraMatrix[14],
                                    viewFarZ.x * rayGen.cameraMatrix[3] + viewFarZ.y * rayGen.cameraMatrix[7] + viewFarZ.z * rayGen.cameraMatrix[11] + rayGen.cameraMatrix[15]);

        ray.direction = normalize(vec3f(rayFarZ.x / rayFarZ.w, rayFarZ.y / rayFarZ.w, rayFarZ.z / rayFarZ.w) - rayGen.cameraLocation);

        // Trace the ray's path
        vec3f colorOut = tracePath(rayGen, ray, prd) * PROGRAM_EXPOSURE_FACTOR;

        // Clamp the output color
        colorOut.x = max(min(colorOut.x, 1.0f), 0.0f);
        colorOut.y = max(min(colorOut.y, 1.0f), 0.0f);
        colorOut.z = max(min(colorOut.z, 1.0f), 0.0f);

        color += colorOut;
    }

    color /= (float)rayGen.samples;

    if (prd.splatTouchesCamera) color = vec3f(1.0f) - color;

    // Assign frame buffer pixel color based on average of all samples
    rayGen.frameBuffer[pixel.x + rayGen.size.x * pixel.y] = make_rgba(color);
}

// Ray hit program
OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
    PerRayData& prd = getPRD<PerRayData>();
    const WorldGeometry& world = getProgramData<WorldGeometry>();

    // Fetch data about the collision surface
    const unsigned int indexPrimitive = optixGetPrimitiveIndex();
    const vec3i index = world.triangles[indexPrimitive];

    // Calculate the normal of the surface
    const vec3f normalSurface = normalize(cross(world.vertices[index.y] - world.vertices[index.x],
                                                world.vertices[index.z] - world.vertices[index.x]));

    const vec3f rayOrigin = optixGetWorldRayOrigin();
    const vec3f rayDir = optixGetWorldRayDirection();

    vec2f uv = optixGetTriangleBarycentrics();
    vec2f textureCoord = (1.0f - uv.x - uv.y) * world.textureCoords[indexPrimitive * 3] +
                         uv.x * world.textureCoords[indexPrimitive * 3 + 1] +
                         uv.y * world.textureCoords[indexPrimitive * 3 + 2];
    vec4f colorTexture = vec4f(tex2D<float4>(world.textures[TEXTURE_DIFFUSE], textureCoord.x, 1.0f - textureCoord.y));

    // Calculate the position of the collision
    prd.hitOrigin = rayOrigin + optixGetRayTmax() * rayDir;

    if (colorTexture.w > prd.random()) { // Ray collided with the material
        // Calculate reflected direction
        vec3f directionReflect = rayDir - 2.0f * dot(rayDir, normalSurface) * normalSurface;
        if(prd.random() > MATERIAL_REFLECTIVITY){ // Scattering for lambertians
            directionReflect = normalSurface + randomUnitSphere(prd.random);
        }

        // Assign final ray data based on all the above calculations
        prd.bounceDirection = directionReflect;

        prd.color = vec3f(colorTexture.x, colorTexture.y, colorTexture.z);
        prd.reflected = true;
    } else { // Ray passed through the material
        prd.bounceDirection = rayDir;
        prd.color = vec3f(1.0f);
    }

    prd.shouldTerminate = false;
}

// Ray miss program
OPTIX_MISS_PROGRAM(miss)() {
    PerRayData& prd = getPRD<PerRayData>();

    vec3f rayDirNormal = normalize(vec3f(optixGetWorldRayDirection()));

    // Generic white/gray sky color function TODO add support for sky sphere textures or similar
    prd.color = vec3f(min(1.0f, 1.0f + rayDirNormal.y));

    prd.shouldTerminate = true;
}
