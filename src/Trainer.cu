#include "hip/hip_runtime.h"
#include <cmath>

#include "ui/UiPanelInput.h"

#include "Trainer.cuh"
#include "TruthCameras.h"

#include <rasterizer.h>
#include <diff-gaussian-rasterization/third_party/glm/glm/gtc/type_ptr.hpp>

__global__ void convertIntKernel(const float* source, uint32_t* frameBuffer, int step, int w, int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        frameBuffer[y * w + x] =
                (min(255, max(0, (int)(source[(y * w + x)] * 256.0f))) << 0) +
                (min(255, max(0, (int)(source[(y * w + x) + 1 * w * h] * 256.0f))) << 8) +
                (min(255, max(0, (int)(source[(y * w + x) + 2 * w * h] * 256.0f))) << 16) +
                (min(255, max(0, (int)(source[(y * w + x) + 3 * w * h] * 256.0f))) << 24);
    }
}

Trainer::Trainer() {
    hipMalloc(&devBackground, 3 * sizeof(float));
    hipMalloc(&devMatView, 16 * sizeof(float));
    hipMalloc(&devMatProjView, 16 * sizeof(float));
    hipMalloc(&devCameraLocation, 3 * sizeof(float));
    hipMalloc(&devRasterized, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 4 * sizeof(float));

    model = new ModelSplats(10, 1, 4);
    /*model->pushBack({0.0f, 0.0f, 0.0f}, {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {1.0f, 1.0f, 1.0f},
                    1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));*/
    model->pushBack({0.0f, 4.0f, 0.0f}, {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {1.0f, 1.0f, 0.1f},
                    1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));

    model->pushBack({0.0f, 0.0f, 0.0f}, {0.0f, 1.0f, 1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {1.0f, 1.0f, 2.0f},
                    1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));
}

Trainer::~Trainer() {
    hipFree(devBackground);
    hipFree(devMatView);
    hipFree(devMatProjView);
    hipFree(devCameraLocation);
    hipFree(devRasterized);

    delete model;
}

void Trainer::render(uint32_t* frameBuffer, TruthCameras& cameras) {
    model->deviceBuffer();

    std::vector<float> background = {0.0f, 0.0f, 0.0f};
    hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matView = -glm::lookAt(TruthCameras::toGlmVec(cameras.getActiveCamera().location),
                                     TruthCameras::toGlmVec(cameras.getActiveCamera().target), {0.0f, 1.0f, 0.0f});
    hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matProjView = glm::perspective(glm::radians(cameras.getActiveCamera().degFovY),
                                             (float)RENDER_RESOLUTION_X / (float)RENDER_RESOLUTION_Y, 0.1f, 100.0f) * matView;
    hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devCameraLocation, &cameras.getActiveCamera().location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

    char* geomBuffer;
    char* binningBuffer;
    char* imgBuffer;

    CudaRasterizer::Rasterizer::forward(
            [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
            [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
            [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
            model->count,
            model->shDegree,
            model->shCoeffs,
            devBackground,
            RENDER_RESOLUTION_X,
            RENDER_RESOLUTION_Y,
            model->devLocations,
            model->devShs,
            nullptr,
            model->devOpacities,
            model->devScales,
            1.0f,
            model->devRotations,
            nullptr,
            devMatView,
            devMatProjView,
            devCameraLocation,
            tan(glm::radians(cameras.getActiveCamera().degFovX) * 0.5f),
            tan(glm::radians(cameras.getActiveCamera().degFovY) * 0.5f),
            false,
            devRasterized,
            nullptr,
            true
            );

    convertIntKernel<<<256, 256>>>(devRasterized, frameBuffer, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

    hipFree(geomBuffer);
    hipFree(binningBuffer);
    hipFree(imgBuffer);
}
