#include "hip/hip_runtime.h"
#include <unordered_set>
#include <cmath>
#include <fstream>

#include "ui/UiPanelInput.h"

#include "Trainer.cuh"
#include "TruthCameras.h"

#include <rasterizer.h>
#include <diff-gaussian-rasterization/third_party/glm/glm/gtc/type_ptr.hpp>

__global__ void convertIntKernel(const float* source, uint32_t* frameBuffer, int step, int w, int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        frameBuffer[y * w + x] =
                (min(255, max(0, (int)(source[(y * w + x)] * 256.0f))) << 0) +
                (min(255, max(0, (int)(source[(y * w + x) + w * h] * 256.0f))) << 8) +
                (min(255, max(0, (int)(source[(y * w + x) + 2 * w * h] * 256.0f))) << 16) + (0xFF << 24);
    }
}

__global__ void lossKernel(const uint32_t* truth, const float* rasterized, float* loss, int step, int w, int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        uint32_t truthRgba = truth[y * w + x];

        loss[y * w + x] = ((float)(truthRgba & 0xFF) / 255.0f) - rasterized[y * w + x];
        loss[(y * w + x) + w * h] = ((float)((truthRgba >> 8) & 0xFF) / 255.0f) - rasterized[(y * w + x) + w * h];
        loss[(y * w + x) + 2 * w * h] = ((float)((truthRgba >> 16) & 0xFF) / 255.0f) - rasterized[(y * w + x) + 2 * w * h];
    }
}

__global__ void gradientSumKernel(float* avgLocations, float* avgShs, float* avgScales, float* avgOpacities, float* avgRotations,
                                      const float* locations, const float* shs, const float* scales, const float* opacities, const float* rotations,
                                      float samples, int shCoeffs, int step, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += step){
        avgLocations[i * 3] += locations[i * 3] / samples;
        avgLocations[i * 3 + 1] += locations[i * 3 + 1] / samples;
        avgLocations[i * 3 + 2] += locations[i * 3 + 2] / samples;

        for(int s = 0; s < shCoeffs; s++) {
            avgShs[(i * 3 * shCoeffs) + 3 * s] += shs[(i * 3 * shCoeffs) + 3 * s] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 1] += shs[(i * 3 * shCoeffs) + 3 * s + 1] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 2] += shs[(i * 3 * shCoeffs) + 3 * s + 2] / samples;
        }

        avgScales[i * 3] += scales[i * 3] / samples;
        avgScales[i * 3 + 1] += scales[i * 3 + 1] / samples;
        avgScales[i * 3 + 2] += scales[i * 3 + 2] / samples;

        avgOpacities[i] += opacities[i] / samples;

        avgRotations[i * 4] += rotations[i * 4] / samples;
        avgRotations[i * 4 + 1] += rotations[i * 4 + 1] / samples;
        avgRotations[i * 4 + 2] += rotations[i * 4 + 2] / samples;
        avgRotations[i * 4 + 3] += rotations[i * 4 + 3] / samples;
    }
}

__global__ void locationVarianceKernel(float* varLocations, const float* gradLocations, float samples, int step, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += step){
        varLocations[i] += sqrtf((gradLocations[i * 3] * gradLocations[i * 3]) +
                                 (gradLocations[i * 3 + 1] * gradLocations[i * 3 + 1]) +
                                 (gradLocations[i * 3 + 2] * gradLocations[i * 3 + 2])) / samples;
    }
}

Trainer::Trainer() {
    hipMalloc(&devBackground, 3 * sizeof(float));
    hipMalloc(&devMatView, 16 * sizeof(float));
    hipMalloc(&devMatProjView, 16 * sizeof(float));
    hipMalloc(&devCameraLocation, 3 * sizeof(float));
    hipMalloc(&devRasterized, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

    // Scene-sized cube initialization
    model = new ModelSplats(1000000, 1, 4);

    static const float dim = 4.0f;
    static const float step = 0.5f;

    for(float x = -dim; x <= dim; x += step){
        for(float y = -dim; y <= dim; y += step){
            for(float z = -dim; z <= dim; z += step){
                model->pushBack({x, y, z}, {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f}, {step * 0.1f, step * 0.1f, step * 0.1f},
                                1.0f, glm::angleAxis(0.0f, glm::vec3(0.0f, 1.0f, 0.0f)));
                //model->pushBack({x, y, z}, {0.0f, 0.0f, 0.0f}, {step * 0.1f, step * 0.1f, step * 0.1f},
                //                1.0f, glm::angleAxis(0.0f, glm::vec3(0.0f, 1.0f, 0.0f)));
            }
        }
    }

    //model->pushBack({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f}, {0.1f, 0.1f, 0.3f},
    //                1.0f, glm::angleAxis(45.0f, glm::vec3(0.0f, 1.0f, 0.0f)));
}

Trainer::~Trainer() {
    hipFree(devBackground);
    hipFree(devMatView);
    hipFree(devMatProjView);
    hipFree(devCameraLocation);
    hipFree(devRasterized);

    hipFree(devVarLocations);
    hipFree(devAvgGradLocations);
    hipFree(devAvgGradShs);
    hipFree(devAvgGradScales);
    hipFree(devAvgGradOpacities);
    hipFree(devAvgGradRotations);

    hipFree(devLossPixels);

    hipFree(devGradLocations);
    hipFree(devGradShs);
    hipFree(devGradScales);
    hipFree(devGradOpacities);
    hipFree(devGradRotations);

    hipFree(devGradMean2D);
    hipFree(devGradConic);
    hipFree(devGradColor);
    hipFree(devGradCov3D);

    delete[] varLocations;
    delete[] avgGradLocations;
    delete[] avgGradShs;
    delete[] avgGradScales;
    delete[] avgGradOpacities;
    delete[] avgGradRotations;

    delete model;

    for (uint32_t* frameBuffer : truthFrameBuffersW) hipFree(frameBuffer);
    for (uint32_t* frameBuffer : truthFrameBuffersB) hipFree(frameBuffer);
}

void Trainer::render(uint32_t* frameBuffer, const Camera& camera) {
    model->deviceBuffer();

    std::vector<float> background = {0.0f, 0.0f, 0.0f};
    hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matView = -glm::lookAt(TruthCameras::toGlmVec(camera.location),
                                     TruthCameras::toGlmVec(camera.target), {0.0f, 1.0f, 0.0f});
    hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matProjView = glm::perspective(glm::radians(camera.degFovY),
                                             (float)RENDER_RESOLUTION_X / (float)RENDER_RESOLUTION_Y, 0.1f, 100.0f) * matView;
    hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

    char* geomBuffer;
    char* binningBuffer;
    char* imgBuffer;

    CudaRasterizer::Rasterizer::forward(
            [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
            [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
            [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
            model->count,
            model->shDegree,
            model->shCoeffs,
            devBackground,
            RENDER_RESOLUTION_X,
            RENDER_RESOLUTION_Y,
            model->devLocations,
            model->devShs,
            nullptr,
            model->devOpacities,
            model->devScales,
            1.0f,
            model->devRotations,
            nullptr,
            devMatView,
            devMatProjView,
            devCameraLocation,
            tan(glm::radians(camera.degFovX) * 0.5f),
            tan(glm::radians(camera.degFovY) * 0.5f),
            false,
            devRasterized,
            nullptr,
            true);

    convertIntKernel<<<256, 256>>>(devRasterized, frameBuffer, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

    hipFree(geomBuffer);
    hipFree(binningBuffer);
    hipFree(imgBuffer);
}

void Trainer::captureTruths(const TruthCameras& cameras, RtxHost& rtx) {
    for (uint32_t* frameBuffer : truthFrameBuffersW) hipFree(frameBuffer);
    for (uint32_t* frameBuffer : truthFrameBuffersB) hipFree(frameBuffer);
    truthFrameBuffersW.clear();
    truthFrameBuffersB.clear();
    truthCameras.clear();

    for (int i = 0; i < cameras.getCount(); i++) {
        Camera camera = cameras.getCamera(i);

        uint32_t* frameBufferW;
        hipMalloc(&frameBufferW, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * sizeof(uint32_t));
        rtx.render(frameBufferW, camera, {1.0f, 1.0f, 1.0f}, nullptr);
        truthFrameBuffersW.push_back(frameBufferW);

        uint32_t* frameBufferB;
        hipMalloc(&frameBufferB, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * sizeof(uint32_t));
        rtx.render(frameBufferB, camera, {0.0f, 0.0f, 0.0f}, nullptr);
        truthFrameBuffersB.push_back(frameBufferB);

        truthCameras.push_back(camera);
    }
}

void Trainer::train(int iter) {
    for(int i = 0; i < iter; i++) train(false);
}

void Trainer::train(bool densify) {
    assert(truths.size() > 0);

    iterations++;

    model->deviceBuffer();

    bool dirty = model->count != lastCount;
    lastCount = model->count;

    if(dirty) {
        hipFree(devVarLocations);
        hipFree(devAvgGradLocations);
        hipFree(devAvgGradShs);
        hipFree(devAvgGradScales);
        hipFree(devAvgGradOpacities);
        hipFree(devAvgGradRotations);

        hipFree(devLossPixels);

        hipFree(devGradLocations);
        hipFree(devGradShs);
        hipFree(devGradScales);
        hipFree(devGradOpacities);
        hipFree(devGradRotations);

        hipFree(devGradMean2D);
        hipFree(devGradConic);
        hipFree(devGradColor);
        hipFree(devGradCov3D);

        delete[] varLocations;
        delete[] avgGradLocations;
        delete[] avgGradShs;
        delete[] avgGradScales;
        delete[] avgGradOpacities;
        delete[] avgGradRotations;

        hipMalloc(&devVarLocations, model->count * sizeof(float));
        hipMalloc(&devAvgGradLocations, model->count * 3 * sizeof(float));
        hipMalloc(&devAvgGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMalloc(&devAvgGradScales, model->count * 3 * sizeof(float));
        hipMalloc(&devAvgGradOpacities, model->count * sizeof(float));
        hipMalloc(&devAvgGradRotations, model->count * 4 * sizeof(float));

        hipMalloc(&devLossPixels, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

        hipMalloc(&devGradLocations, model->count * 3 * sizeof(float));
        hipMalloc(&devGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMalloc(&devGradScales, model->count * 3 * sizeof(float));
        hipMalloc(&devGradOpacities, model->count * sizeof(float));
        hipMalloc(&devGradRotations, model->count * 4 * sizeof(float));

        hipMalloc(&devGradMean2D, model->count * 3 * sizeof(float));
        hipMalloc(&devGradConic, model->count * 4 * sizeof(float));
        hipMalloc(&devGradColor, model->count * 3 * sizeof(float));
        hipMalloc(&devGradCov3D, model->count * 6 * sizeof(float));

        varLocations = new float[model->count];
        avgGradLocations = new float[model->count * 3];
        avgGradShs = new float[model->count * 3 * model->shCoeffs];
        avgGradScales = new float[model->count * 3];
        avgGradOpacities = new float[model->count];
        avgGradRotations = new float[model->count * 4];
    }

    hipMemset(devVarLocations, 0, model->count * sizeof(float));
    hipMemset(devAvgGradLocations, 0, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
    hipMemset(devAvgGradScales, 0, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradOpacities, 0, model->count * sizeof(float));
    hipMemset(devAvgGradRotations, 0, model->count * 4 * sizeof(float));

    for (int i = 0; i < truthFrameBuffersW.size() * 2; i++) {
        bool backgroundWhite = i < truthFrameBuffersW.size();
        uint32_t* truthFrameBuffer = backgroundWhite ? truthFrameBuffersW[i] : truthFrameBuffersB[i - truthFrameBuffersW.size()];
        Camera camera = truthCameras[i % truthFrameBuffersW.size()];

        std::vector<float> background;
        for(int c = 0; c < 3; c++) background.emplace_back(backgroundWhite ? 1.0f : 0.0f);
        hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

        glm::mat4 matView = camera.getView();
        glm::mat4 matProjView = camera.getProjection() * matView;
        hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

        char* geomBuffer;
        char* binningBuffer;
        char* imgBuffer;

        int countRendered = CudaRasterizer::Rasterizer::forward(
                [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
                [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
                [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
                model->count,
                model->shDegree,
                model->shCoeffs,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devOpacities,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                false,
                devRasterized,
                nullptr,
                true);

        lossKernel<<<256, 256>>>(truthFrameBuffer, devRasterized, devLossPixels, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

        hipMemset(devGradLocations, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMemset(devGradScales, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradOpacities, 0, model->count * sizeof(float));
        hipMemset(devGradRotations, 0, model->count * 4 * sizeof(float));

        hipMemset(devGradMean2D, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradConic, 0, model->count * 4 * sizeof(float));
        hipMemset(devGradColor, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradCov3D, 0, model->count * 6 * sizeof(float));

        CudaRasterizer::Rasterizer::backward(
                model->count,
                model->shDegree,
                model->shCoeffs,
                countRendered,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                nullptr, // TODO pass in from forward rasterizer to increase speed
                geomBuffer,
                binningBuffer,
                imgBuffer,
                devLossPixels,
                devGradMean2D,
                devGradConic,
                devGradOpacities,
                devGradColor,
                devGradLocations,
                devGradCov3D,
                devGradShs,
                devGradScales,
                devGradRotations,
                true);

        gradientSumKernel<<<256, 256>>>(devAvgGradLocations, devAvgGradShs, devAvgGradScales, devAvgGradOpacities, devAvgGradRotations,
                                        devGradLocations, devGradShs, devGradScales, devGradOpacities, devGradRotations,
                                        (float)truthFrameBuffersW.size() * 2.0f, model->shCoeffs, 256 * 256, model->count);

        locationVarianceKernel<<<256, 256>>>(devVarLocations, devGradLocations, (float)truthFrameBuffersW.size() * 2.0f, 256 * 256, model->count);

        hipFree(geomBuffer);
        hipFree(binningBuffer);
        hipFree(imgBuffer);
    }

    hipMemcpy(varLocations, devVarLocations, model->count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(avgGradLocations, devAvgGradLocations, model->count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(avgGradShs, devAvgGradShs, model->count * 3 * model->shCoeffs * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(avgGradScales, devAvgGradScales, model->count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(avgGradOpacities, devAvgGradOpacities, model->count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(avgGradRotations, devAvgGradRotations, model->count * 4 * sizeof(float), hipMemcpyDeviceToHost);

    static const float learningRate = 0.00005f;
    static const float learningRateShs = learningRate * 2.0f;
    static const float learningRateOpacity = learningRate * 2.0f;
    static const float learningRateRotation = learningRate / 2.0f;

    // Apply gradients
    std::unordered_set<int> toSplit, toClone, toRemove;
    for(int i = 0; i < model->count; i++) {
        for(int f = 0; f < 3; f++) {
            model->locations[i * 3 + f] += avgGradLocations[i * 3 + f] * learningRate;
        }
        for(int f  = 0; f < model->shCoeffs * 3; f++) {
            model->shs[i * 3 * model->shCoeffs + f] += avgGradShs[i * 3 * model->shCoeffs + f] * learningRateShs;
        }
        for(int f = 0; f < 3; f++) {
            model->scales[i * 3 + f] += avgGradScales[i * 3 + f] * learningRate;
            model->scales[i * 3 + f] = std::min(0.3f, std::max(0.0f, model->scales[i * 3 + f]));
        }
        model->opacities[i] = std::min(1.0f, std::max(0.0f, model->opacities[i] + avgGradOpacities[i] * learningRateOpacity));
        for(int f = 0; f < 4; f++) {
            model->rotations[i * 4 + f] += avgGradRotations[i * 4 + f] * learningRateRotation;
        }

        if (model->opacities[i] <= 0.005f || glm::length(glm::vec3(model->scales[i * 3], model->scales[i * 3 + 1], model->scales[i * 3 + 2])) < 0.0001f) {
            toRemove.insert(i);
        } else if (varLocations[i] > 2.0f) {
            if (glm::length(glm::vec3(model->scales[i * 3], model->scales[i * 3 + 1], model->scales[i * 3 + 2])) > 0.02f) toSplit.insert(i);
            else toClone.insert(i);
        }
    }

    if(densify) {
        for (int i : toSplit) {
            if (model->count < model->capacity) {
                glm::vec3 locPre(model->locations[i * 3], model->locations[i * 3 + 1], model->locations[i * 3 + 2]);
                glm::vec3 scalePre(model->scales[i * 3], model->scales[i * 3 + 1], model->scales[i * 3 + 2]);
                glm::quat rotPre(model->rotations[i * 4], model->rotations[i * 4 + 1], model->rotations[i * 4 + 2], model->rotations[i * 4 + 3]);

                glm::vec4 scaleOffset4(scalePre.x, scalePre.y, scalePre.z, 1.0f);
                if (scalePre.x > scalePre.y && scalePre.x > scalePre.z) {
                    scaleOffset4 *= glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
                } else if (scalePre.y > scalePre.z) {
                    scaleOffset4 *= glm::vec4(0.0f, 1.0f, 0.0f, 1.0f);
                } else {
                    scaleOffset4 *= glm::vec4(0.0f, 0.0f, 1.0f, 1.0f);
                }

                scaleOffset4 = (glm::mat4)rotPre * scaleOffset4;

                glm::vec3 locNew1 = locPre + glm::vec3(scaleOffset4.x, scaleOffset4.y, scaleOffset4.z) * 0.75f;
                glm::vec3 locNew2 = locPre - glm::vec3(scaleOffset4.x, scaleOffset4.y, scaleOffset4.z) * 0.75f;

                glm::vec3 scaleNew = glm::vec3(model->scales[i * 3], model->scales[i * 3 + 1], model->scales[i * 3 + 2]) / 2.0f;

                int i2 = model->count;
                model->count++;
                model->copy(i2, i); // Create a duplicate splat at the end of the array

                memcpy(&model->locations[i * 3], &locNew1[0], 3 * sizeof(float));
                memcpy(&model->locations[i2 * 3], &locNew2[0], 3 * sizeof(float));
                memcpy(&model->scales[i * 3], &scaleNew[0], 3 * sizeof(float));
                memcpy(&model->scales[i2 * 3], &scaleNew[0], 3 * sizeof(float));
                memcpy(&model->rotations[i * 4], &rotPre[0], 4 * sizeof(float));
                memcpy(&model->rotations[i2 * 4], &rotPre[0], 4 * sizeof(float));
            }
        }

        for (int i : toClone) {
            if (model->count < model->capacity) {
                glm::vec3 loc(model->locations[i * 3], model->locations[i * 3 + 1], model->locations[i * 3 + 2]);
                loc += glm::vec3(avgGradLocations[i * 3] * learningRate,
                                 avgGradLocations[i * 3 + 1] * learningRate,
                                 avgGradLocations[i * 3 + 2] * learningRate);

                int i2 = model->count;
                model->count++;
                model->copy(i2, i); // Create a duplicate splat at the end of the array

                memcpy(&model->locations[i2 * 3], &loc[0], 3 * sizeof(float));
            }
        }

        // Prune small/transparent splats
        if (!toRemove.empty()) {
            int indexPreserved = 0;
            for (int indexScan = 0; indexScan < model->count; indexScan++) {
                if (!toRemove.count(indexScan)) {
                    if (indexPreserved != indexScan) model->copy(indexPreserved, indexScan);
                    indexPreserved++;
                }
            }
            model->count -= (int)toRemove.size();
        }
    }
}
