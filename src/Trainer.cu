#include "hip/hip_runtime.h"
#include <unordered_set>
#include <cmath>
#include <fstream>

#include "ui/UiPanelInput.h"

#include "Trainer.cuh"
#include "TruthCameras.h"
#include "ModelSplatsHost.h"

#include <rasterizer.h>
#include <diff-gaussian-rasterization/third_party/glm/glm/gtc/type_ptr.hpp>

__global__ void imageFloatToInt(const float* source, uint32_t* frameBuffer, const int step, const int w, const int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        frameBuffer[y * w + x] =
                (min(255, max(0, (int)(source[(y * w + x)] * 256.0f))) << 0) +
                (min(255, max(0, (int)(source[(y * w + x) + w * h] * 256.0f))) << 8) +
                (min(255, max(0, (int)(source[(y * w + x) + 2 * w * h] * 256.0f))) << 16) + (0xFF << 24);
    }
}

__global__ void imageIntToLoss(const uint32_t* truth, const float* rasterized, float* loss, const int step, const int w, const int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        uint32_t truthRgba = truth[y * w + x];

        loss[y * w + x] = ((float)(truthRgba & 0xFF) / 255.0f) - rasterized[y * w + x];
        loss[(y * w + x) + w * h] = ((float)((truthRgba >> 8) & 0xFF) / 255.0f) - rasterized[(y * w + x) + w * h];
        loss[(y * w + x) + 2 * w * h] = ((float)((truthRgba >> 16) & 0xFF) / 255.0f) - rasterized[(y * w + x) + 2 * w * h];
    }
}

__global__ void accumulateGradients(float* avgLocations, float* avgShs, float* avgScales, float* avgOpacities, float* avgRotations,
                                    const float* locations, const float* shs, const float* scales, const float* opacities, const float* rotations,
                                    const float samples, const int shCoeffs, const int step, const int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += step){
        avgLocations[i * 3] += locations[i * 3] / samples;
        avgLocations[i * 3 + 1] += locations[i * 3 + 1] / samples;
        avgLocations[i * 3 + 2] += locations[i * 3 + 2] / samples;

        for(int s = 0; s < shCoeffs; s++) {
            avgShs[(i * 3 * shCoeffs) + 3 * s] += shs[(i * 3 * shCoeffs) + 3 * s] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 1] += shs[(i * 3 * shCoeffs) + 3 * s + 1] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 2] += shs[(i * 3 * shCoeffs) + 3 * s + 2] / samples;
        }

        avgScales[i * 3] += scales[i * 3] / samples;
        avgScales[i * 3 + 1] += scales[i * 3 + 1] / samples;
        avgScales[i * 3 + 2] += scales[i * 3 + 2] / samples;

        avgOpacities[i] += opacities[i] / samples;

        avgRotations[i * 4] += rotations[i * 4] / samples;
        avgRotations[i * 4 + 1] += rotations[i * 4 + 1] / samples;
        avgRotations[i * 4 + 2] += rotations[i * 4 + 2] / samples;
        avgRotations[i * 4 + 3] += rotations[i * 4 + 3] / samples;
    }
}

__global__ void accumulateVariance(float* varLocations, const float* gradLocations, const float samples, const int step, const int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += step){
        varLocations[i] += sqrtf((gradLocations[i * 3] * gradLocations[i * 3]) +
                                 (gradLocations[i * 3 + 1] * gradLocations[i * 3 + 1]) +
                                 (gradLocations[i * 3 + 2] * gradLocations[i * 3 + 2])) / samples;
    }
}

__global__ void applyGradients(float* locations, float* shs, float* scales, float* opacities, float* rotations,
                               const float* gradLocations, const float* gradShs, const float* gradScales, const float* gradOpacities, const float* gradRotations,
                               const float lr, const float lrSh, const float lrOpacity, const float lrRotation,
                               const int shCoeffs, const int step, const int count) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += step){
        for(int f = 0; f < 3; f++) {
            locations[i * 3 + f] += gradLocations[i * 3 + f] * lr;
        }
        for(int f  = 0; f < shCoeffs * 3; f++) {
            shs[i * 3 * shCoeffs + f] += gradShs[i * 3 * shCoeffs + f] * lrSh;
        }
        for(int f = 0; f < 3; f++) {
            scales[i * 3 + f] += gradScales[i * 3 + f] * lr;
            scales[i * 3 + f] = min(0.3f, max(0.0f, scales[i * 3 + f]));
        }
        opacities[i] = min(1.0f, max(0.0f, opacities[i] + gradOpacities[i] * lrOpacity));
        for(int f = 0; f < 4; f++) {
            rotations[i * 4 + f] += gradRotations[i * 4 + f] * lrRotation;
        }
    }
}

Trainer::Trainer() {
    hipMalloc(&devBackground, 3 * sizeof(float));
    hipMalloc(&devMatView, 16 * sizeof(float));
    hipMalloc(&devMatProjView, 16 * sizeof(float));
    hipMalloc(&devCameraLocation, 3 * sizeof(float));
    hipMalloc(&devRasterized, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

    // Scene-sized cube initialization
    ModelSplatsHost modelHost(1000000, 1, 4);

    static const float dim = 4.0f;
    static const float step = 0.5f;

    for(float x = -dim; x <= dim; x += step){
        for(float y = -dim; y <= dim; y += step){
            for(float z = -dim; z <= dim; z += step){
                modelHost.pushBack({x, y, z}, {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f}, {step * 0.1f, step * 0.1f, step * 0.1f},
                                1.0f, glm::angleAxis(0.0f, glm::vec3(0.0f, 1.0f, 0.0f)));
                //modelHost.pushBack({x, y, z}, {0.0f, 0.0f, 0.0f}, {step * 0.1f, step * 0.1f, step * 0.1f},
                //                1.0f, glm::angleAxis(0.0f, glm::vec3(0.0f, 1.0f, 0.0f)));
            }
        }
    }

    //modelHost.pushBack({0.0f, 0.0f, 0.0f}, {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f}, {0.1f, 0.1f, 0.3f},
    //                1.0f, glm::angleAxis(45.0f, glm::vec3(0.0f, 1.0f, 0.0f)));

    model = new ModelSplatsDevice(modelHost);
}

Trainer::~Trainer() {
    hipFree(devBackground);
    hipFree(devMatView);
    hipFree(devMatProjView);
    hipFree(devCameraLocation);
    hipFree(devRasterized);

    hipFree(devVarLocations);
    hipFree(devAvgGradLocations);
    hipFree(devAvgGradShs);
    hipFree(devAvgGradScales);
    hipFree(devAvgGradOpacities);
    hipFree(devAvgGradRotations);

    hipFree(devLossPixels);

    hipFree(devGradLocations);
    hipFree(devGradShs);
    hipFree(devGradScales);
    hipFree(devGradOpacities);
    hipFree(devGradRotations);

    hipFree(devGradMean2D);
    hipFree(devGradConic);
    hipFree(devGradColor);
    hipFree(devGradCov3D);

    delete model;

    for (uint32_t* frameBuffer : truthFrameBuffersW) hipFree(frameBuffer);
    for (uint32_t* frameBuffer : truthFrameBuffersB) hipFree(frameBuffer);
}

void Trainer::render(uint32_t* frameBuffer, const Camera& camera) {
    std::vector<float> background = {0.0f, 0.0f, 0.0f};
    hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matView = camera.getView();
    glm::mat4 matProjView = camera.getProjection() * matView;
    hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

    char* geomBuffer;
    char* binningBuffer;
    char* imgBuffer;

    CudaRasterizer::Rasterizer::forward(
            [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
            [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
            [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
            model->count,
            model->shDegree,
            model->shCoeffs,
            devBackground,
            RENDER_RESOLUTION_X,
            RENDER_RESOLUTION_Y,
            model->devLocations,
            model->devShs,
            nullptr,
            model->devOpacities,
            model->devScales,
            1.0f,
            model->devRotations,
            nullptr,
            devMatView,
            devMatProjView,
            devCameraLocation,
            tan(glm::radians(camera.degFovX) * 0.5f),
            tan(glm::radians(camera.degFovY) * 0.5f),
            false,
            devRasterized,
            nullptr,
            true);

    imageFloatToInt<<<256, 256>>>(devRasterized, frameBuffer, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

    hipFree(geomBuffer);
    hipFree(binningBuffer);
    hipFree(imgBuffer);
}

void Trainer::captureTruths(const TruthCameras& cameras, RtxHost& rtx) {
    if (lastTruthCount != cameras.getCount()) {
        lastTruthCount = cameras.getCount();

        for (uint32_t* frameBuffer : truthFrameBuffersW) hipFree(frameBuffer);
        for (uint32_t* frameBuffer : truthFrameBuffersB) hipFree(frameBuffer);
        truthFrameBuffersW.clear();
        truthFrameBuffersB.clear();

        truthCameras.clear();

        for (int i = 0; i < cameras.getCount(); i++) {
            Camera camera = cameras.getCamera(i);

            uint32_t* frameBufferW;
            hipMalloc(&frameBufferW, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * sizeof(uint32_t));
            rtx.render(frameBufferW, camera, {1.0f, 1.0f, 1.0f}, nullptr);
            truthFrameBuffersW.push_back(frameBufferW);

            uint32_t* frameBufferB;
            hipMalloc(&frameBufferB, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * sizeof(uint32_t));
            rtx.render(frameBufferB, camera, {0.0f, 0.0f, 0.0f}, nullptr);
            truthFrameBuffersB.push_back(frameBufferB);

            truthCameras.push_back(camera);
        }
    } else {
        for (int i = 0; i < cameras.getCount(); i++) {
            Camera camera = cameras.getCamera(i);
            rtx.render(truthFrameBuffersW.at(i), camera, {1.0f, 1.0f, 1.0f}, nullptr);
            rtx.render(truthFrameBuffersB.at(i), camera, {0.0f, 0.0f, 0.0f}, nullptr);
            truthCameras.at(i) = camera;
        }
    }
}

void Trainer::train(int iter) {
    for(int i = 0; i < iter; i++) train(false);
}

void Trainer::train(bool densify) {
    assert(truths.size() > 0);

    iterations++;

    if (model->count != lastCount) {
        hipFree(devVarLocations);
        hipFree(devAvgGradLocations);
        hipFree(devAvgGradShs);
        hipFree(devAvgGradScales);
        hipFree(devAvgGradOpacities);
        hipFree(devAvgGradRotations);

        hipFree(devLossPixels);

        hipFree(devGradLocations);
        hipFree(devGradShs);
        hipFree(devGradScales);
        hipFree(devGradOpacities);
        hipFree(devGradRotations);

        hipFree(devGradMean2D);
        hipFree(devGradConic);
        hipFree(devGradColor);
        hipFree(devGradCov3D);

        hipMalloc(&devVarLocations, model->count * sizeof(float));
        hipMalloc(&devAvgGradLocations, model->count * 3 * sizeof(float));
        hipMalloc(&devAvgGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMalloc(&devAvgGradScales, model->count * 3 * sizeof(float));
        hipMalloc(&devAvgGradOpacities, model->count * sizeof(float));
        hipMalloc(&devAvgGradRotations, model->count * 4 * sizeof(float));

        hipMalloc(&devLossPixels, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

        hipMalloc(&devGradLocations, model->count * 3 * sizeof(float));
        hipMalloc(&devGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMalloc(&devGradScales, model->count * 3 * sizeof(float));
        hipMalloc(&devGradOpacities, model->count * sizeof(float));
        hipMalloc(&devGradRotations, model->count * 4 * sizeof(float));

        hipMalloc(&devGradMean2D, model->count * 3 * sizeof(float));
        hipMalloc(&devGradConic, model->count * 4 * sizeof(float));
        hipMalloc(&devGradColor, model->count * 3 * sizeof(float));
        hipMalloc(&devGradCov3D, model->count * 6 * sizeof(float));

        lastCount = model->count;
    }

    hipMemset(devVarLocations, 0, model->count * sizeof(float));
    hipMemset(devAvgGradLocations, 0, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
    hipMemset(devAvgGradScales, 0, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradOpacities, 0, model->count * sizeof(float));
    hipMemset(devAvgGradRotations, 0, model->count * 4 * sizeof(float));

    for (int i = 0; i < truthFrameBuffersW.size() * 2; i++) {
        bool backgroundWhite = i < truthFrameBuffersW.size();
        uint32_t* truthFrameBuffer = backgroundWhite ? truthFrameBuffersW[i] : truthFrameBuffersB[i - truthFrameBuffersW.size()];
        Camera camera = truthCameras[i % truthFrameBuffersW.size()];

        std::vector<float> background;
        for(int c = 0; c < 3; c++) background.emplace_back(backgroundWhite ? 1.0f : 0.0f);
        hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

        glm::mat4 matView = camera.getView();
        glm::mat4 matProjView = camera.getProjection() * matView;
        hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

        char* geomBuffer;
        char* binningBuffer;
        char* imgBuffer;

        int countRendered = CudaRasterizer::Rasterizer::forward(
                [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
                [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
                [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
                model->count,
                model->shDegree,
                model->shCoeffs,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devOpacities,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                false,
                devRasterized,
                nullptr,
                true);

        imageIntToLoss<<<256, 256>>>(truthFrameBuffer, devRasterized, devLossPixels, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

        hipMemset(devGradLocations, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMemset(devGradScales, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradOpacities, 0, model->count * sizeof(float));
        hipMemset(devGradRotations, 0, model->count * 4 * sizeof(float));

        hipMemset(devGradMean2D, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradConic, 0, model->count * 4 * sizeof(float));
        hipMemset(devGradColor, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradCov3D, 0, model->count * 6 * sizeof(float));

        CudaRasterizer::Rasterizer::backward(
                model->count,
                model->shDegree,
                model->shCoeffs,
                countRendered,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                nullptr, // TODO pass in from forward rasterizer to increase speed
                geomBuffer,
                binningBuffer,
                imgBuffer,
                devLossPixels,
                devGradMean2D,
                devGradConic,
                devGradOpacities,
                devGradColor,
                devGradLocations,
                devGradCov3D,
                devGradShs,
                devGradScales,
                devGradRotations,
                true);

        accumulateGradients<<<256, 256>>>(devAvgGradLocations, devAvgGradShs, devAvgGradScales, devAvgGradOpacities,devAvgGradRotations,
                                          devGradLocations, devGradShs, devGradScales, devGradOpacities, devGradRotations,
                                          (float)truthFrameBuffersW.size() * 2.0f, model->shCoeffs, 256 * 256, model->count);

        accumulateVariance<<<256, 256>>>(devVarLocations, devGradLocations,
                                         (float)truthFrameBuffersW.size() * 2.0f, 256 * 256, model->count);

        hipFree(geomBuffer);
        hipFree(binningBuffer);
        hipFree(imgBuffer);
    }

    static const float learningRate = 0.00005f;
    static const float learningRateShs = learningRate * 2.0f;
    static const float learningRateOpacity = learningRate * 2.0f;
    static const float learningRateRotation = learningRate / 2.0f;

    applyGradients<<<256, 256>>>(model->devLocations, model->devShs, model->devScales, model->devOpacities, model->devRotations,
                                 devAvgGradLocations, devAvgGradShs, devAvgGradScales, devAvgGradOpacities, devAvgGradRotations,
                                 learningRate, learningRateShs, learningRateOpacity, learningRateRotation,
                                 model->shCoeffs, 256 * 256, model->count);


    if(densify) {
        // This is a densify iteration, so transfer the model back to host memory. The complex splitting/pruning
        // operations easier here than on the GPU! This is slower, but it's only about 1/100 iterations so the tradeoff
        // is worth it.
        ModelSplatsHost modelHost(*model);


        float* varLocations = new float[model->count];
        float* gradLocations = new float[model->count * 3];
        hipMemcpy(varLocations, devVarLocations, model->count * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(gradLocations, devAvgGradLocations, model->count * 3 * sizeof(float), hipMemcpyDeviceToHost);

        std::unordered_set<int> toSplit, toClone, toRemove;
        for(int i = 0; i < modelHost.count; i++) {
            if (modelHost.opacities[i] <= 0.005f || glm::length(glm::vec3(modelHost.scales[i * 3], modelHost.scales[i * 3 + 1], modelHost.scales[i * 3 + 2])) < 0.0001f) {
                toRemove.insert(i);
            } else if (varLocations[i] > 2.0f) {
                if (glm::length(glm::vec3(modelHost.scales[i * 3], modelHost.scales[i * 3 + 1], modelHost.scales[i * 3 + 2])) > 0.02f) toSplit.insert(i);
                else toClone.insert(i);
            }
        }

        for (int i : toSplit) {
            if (modelHost.count < modelHost.capacity) {
                glm::vec3 locPre(modelHost.locations[i * 3], modelHost.locations[i * 3 + 1], modelHost.locations[i * 3 + 2]);
                glm::vec3 scalePre(modelHost.scales[i * 3], modelHost.scales[i * 3 + 1], modelHost.scales[i * 3 + 2]);
                glm::quat rotPre(modelHost.rotations[i * 4], modelHost.rotations[i * 4 + 1], modelHost.rotations[i * 4 + 2], modelHost.rotations[i * 4 + 3]);

                glm::vec4 scaleOffset4(scalePre.x, scalePre.y, scalePre.z, 1.0f);
                if (scalePre.x > scalePre.y && scalePre.x > scalePre.z) {
                    scaleOffset4 *= glm::vec4(1.0f, 0.0f, 0.0f, 1.0f);
                } else if (scalePre.y > scalePre.z) {
                    scaleOffset4 *= glm::vec4(0.0f, 1.0f, 0.0f, 1.0f);
                } else {
                    scaleOffset4 *= glm::vec4(0.0f, 0.0f, 1.0f, 1.0f);
                }

                scaleOffset4 = (glm::mat4)rotPre * scaleOffset4;

                glm::vec3 locNew1 = locPre + glm::vec3(scaleOffset4.x, scaleOffset4.y, scaleOffset4.z) * 0.75f;
                glm::vec3 locNew2 = locPre - glm::vec3(scaleOffset4.x, scaleOffset4.y, scaleOffset4.z) * 0.75f;

                glm::vec3 scaleNew = glm::vec3(modelHost.scales[i * 3], modelHost.scales[i * 3 + 1], modelHost.scales[i * 3 + 2]) / 2.0f;

                int i2 = modelHost.count;
                modelHost.count++;
                modelHost.copy(i2, i); // Create a duplicate splat at the end of the array

                memcpy(&modelHost.locations[i * 3], &locNew1[0], 3 * sizeof(float));
                memcpy(&modelHost.locations[i2 * 3], &locNew2[0], 3 * sizeof(float));
                memcpy(&modelHost.scales[i * 3], &scaleNew[0], 3 * sizeof(float));
                memcpy(&modelHost.scales[i2 * 3], &scaleNew[0], 3 * sizeof(float));
                memcpy(&modelHost.rotations[i * 4], &rotPre[0], 4 * sizeof(float));
                memcpy(&modelHost.rotations[i2 * 4], &rotPre[0], 4 * sizeof(float));
            }
        }

        for (int i : toClone) {
            if (modelHost.count < modelHost.capacity) {
                glm::vec3 loc(modelHost.locations[i * 3], modelHost.locations[i * 3 + 1], modelHost.locations[i * 3 + 2]);
                loc += glm::vec3(gradLocations[i * 3] * learningRate,
                                 gradLocations[i * 3 + 1] * learningRate,
                                 gradLocations[i * 3 + 2] * learningRate);

                int i2 = modelHost.count;
                modelHost.count++;
                modelHost.copy(i2, i); // Create a duplicate splat at the end of the array

                memcpy(&modelHost.locations[i2 * 3], &loc[0], 3 * sizeof(float));
            }
        }

        // Prune small/transparent splats
        if (!toRemove.empty()) {
            int indexPreserved = 0;
            for (int indexScan = 0; indexScan < modelHost.count; indexScan++) {
                if (!toRemove.count(indexScan)) {
                    if (indexPreserved != indexScan) modelHost.copy(indexPreserved, indexScan);
                    indexPreserved++;
                }
            }
            modelHost.count -= (int)toRemove.size();
        }

        delete[] varLocations;
        delete[] gradLocations;

        delete model;
        model = new ModelSplatsDevice(modelHost);
    }
}
