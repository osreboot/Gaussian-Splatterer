#include "hip/hip_runtime.h"
#include <cmath>

#include "ui/UiPanelInput.h"

#include "Trainer.cuh"
#include "TruthCameras.h"

#include <rasterizer.h>
#include <diff-gaussian-rasterization/third_party/glm/glm/gtc/type_ptr.hpp>

__global__ void convertIntKernel(const float* source, uint32_t* frameBuffer, int step, int w, int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        frameBuffer[y * w + x] =
                (min(255, max(0, (int)(source[(y * w + x)] * 256.0f))) << 0) +
                (min(255, max(0, (int)(source[(y * w + x) + w * h] * 256.0f))) << 8) +
                (min(255, max(0, (int)(source[(y * w + x) + 2 * w * h] * 256.0f))) << 16) + (0xFF << 24);
    }
}

__global__ void lossKernel(const uint32_t* truth, const float* rasterized, float* loss, int step, int w, int h) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < w * h; i += step){
        int x = i % w;
        int y = i / w;

        uint32_t truthRgba = truth[y * w + x];

        loss[y * w + x] =
                (((float)(truthRgba & 0xFF) / 255.0f) - rasterized[y * w + x]) / 255.0f;
        loss[(y * w + x) + w * h] =
                (((float)((truthRgba >> 8) & 0xFF) / 255.0f) - rasterized[(y * w + x) + w * h]) / 255.0f;
        loss[(y * w + x) + 2 * w * h] =
                (((float)((truthRgba >> 16) & 0xFF) / 255.0f) - rasterized[(y * w + x) + 2 * w * h]) / 255.0f;
    }
}

__global__ void gradientSumKernel(float* avgLocations, float* avgShs, float* avgScales, float* avgOpacities, float* avgRotations,
                                      float* locations, float* shs, float* scales, float* opacities, float* rotations,
                                      float samples, int shCoeffs, int step, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += step){
        avgLocations[i * 3] += locations[i * 3] / samples;
        avgLocations[i * 3 + 1] += locations[i * 3 + 1] / samples;
        avgLocations[i * 3 + 2] += locations[i * 3 + 2] / samples;

        for(int s = 0; s < shCoeffs; s++) {
            avgShs[(i * 3 * shCoeffs) + 3 * s] += shs[(i * 3 * shCoeffs) + 3 * s] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 1] += shs[(i * 3 * shCoeffs) + 3 * s + 1] / samples;
            avgShs[(i * 3 * shCoeffs) + 3 * s + 2] += shs[(i * 3 * shCoeffs) + 3 * s + 2] / samples;
        }

        avgScales[i * 3] += scales[i * 3] / samples;
        avgScales[i * 3 + 1] += scales[i * 3 + 1] / samples;
        avgScales[i * 3 + 2] += scales[i * 3 + 2] / samples;

        avgOpacities[i] += opacities[i] / samples;

        avgRotations[i * 4] += rotations[i * 4] / samples;
        avgRotations[i * 4 + 1] += rotations[i * 4 + 1] / samples;
        avgRotations[i * 4 + 2] += rotations[i * 4 + 2] / samples;
        avgRotations[i * 4 + 3] += rotations[i * 4 + 3] / samples;
    }
}

Trainer::Trainer() {
    hipMalloc(&devBackground, 3 * sizeof(float));
    hipMalloc(&devMatView, 16 * sizeof(float));
    hipMalloc(&devMatProjView, 16 * sizeof(float));
    hipMalloc(&devCameraLocation, 3 * sizeof(float));
    hipMalloc(&devRasterized, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

    model = new ModelSplats(1000, 1, 4);

    for(int x = 0; x < 5; x++){
        for(int y = 0; y < 5; y++){
            for(int z = 0; z < 5; z++){
                model->pushBack({(float)x - 2.0f, (float)y - 2.0f, (float)z - 2.0f},
                                {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {0.2f, 0.2f, 0.2f},
                                1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));
            }
        }
    }

    /*
    model->pushBack({0.0f, 4.0f, 0.0f}, {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {1.0f, 1.0f, 0.1f},
                    1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));

    model->pushBack({0.0f, 0.0f, 0.0f}, {0.0f, 1.0f, 1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f}, {1.0f, 1.0f, 2.0f},
                    1.0f, glm::angleAxis(glm::radians(0.0f), glm::vec3(0.0f, 1.0f, 0.0f)));*/
}

Trainer::~Trainer() {
    hipFree(devBackground);
    hipFree(devMatView);
    hipFree(devMatProjView);
    hipFree(devCameraLocation);
    hipFree(devRasterized);

    delete model;

    for (uint32_t* frameBuffer : truthFrameBuffers) {
        hipFree(frameBuffer);
    }
}

void Trainer::render(uint32_t* frameBuffer, const Camera& camera) {
    model->deviceBuffer();

    std::vector<float> background = {0.0f, 0.0f, 0.0f};
    hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matView = -glm::lookAt(TruthCameras::toGlmVec(camera.location),
                                     TruthCameras::toGlmVec(camera.target), {0.0f, 1.0f, 0.0f});
    hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);

    glm::mat4 matProjView = glm::perspective(glm::radians(camera.degFovY),
                                             (float)RENDER_RESOLUTION_X / (float)RENDER_RESOLUTION_Y, 0.1f, 100.0f) * matView;
    hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

    char* geomBuffer;
    char* binningBuffer;
    char* imgBuffer;

    CudaRasterizer::Rasterizer::forward(
            [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
            [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
            [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
            model->count,
            model->shDegree,
            model->shCoeffs,
            devBackground,
            RENDER_RESOLUTION_X,
            RENDER_RESOLUTION_Y,
            model->devLocations,
            model->devShs,
            nullptr,
            model->devOpacities,
            model->devScales,
            1.0f,
            model->devRotations,
            nullptr,
            devMatView,
            devMatProjView,
            devCameraLocation,
            tan(glm::radians(camera.degFovX) * 0.5f),
            tan(glm::radians(camera.degFovY) * 0.5f),
            false,
            devRasterized,
            nullptr,
            true);

    convertIntKernel<<<256, 256>>>(devRasterized, frameBuffer, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

    hipFree(geomBuffer);
    hipFree(binningBuffer);
    hipFree(imgBuffer);
}

void Trainer::captureTruths(const TruthCameras& cameras, RtxHost& rtx) {
    for (uint32_t* frameBuffer : truthFrameBuffers) {
        hipFree(frameBuffer);
    }
    truthFrameBuffers.clear();
    truthCameras.clear();

    for (int i = 0; i < cameras.getCount(); i++) {
        uint32_t* frameBuffer;
        Camera camera = cameras.getCamera(i);
        hipMalloc(&frameBuffer, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * sizeof(uint32_t));
        rtx.render(frameBuffer, camera, nullptr);
        truthFrameBuffers.push_back(frameBuffer);
        truthCameras.push_back(camera);
    }
}

void Trainer::train(int iterations) {
    for(int i = 0; i < iterations; i++) train(false);
}

void Trainer::train(bool densify) {
    assert(truths.size() > 0);

    model->deviceBuffer();

    float* devAvgGradLocations;
    hipMalloc(&devAvgGradLocations, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradLocations, 0, model->count * 3 * sizeof(float));
    float* devAvgGradShs;
    hipMalloc(&devAvgGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
    hipMemset(devAvgGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
    float* devAvgGradScales;
    hipMalloc(&devAvgGradScales, model->count * 3 * sizeof(float));
    hipMemset(devAvgGradScales, 0, model->count * 3 * sizeof(float));
    float* devAvgGradOpacities;
    hipMalloc(&devAvgGradOpacities, model->count * sizeof(float));
    hipMemset(devAvgGradOpacities, 0, model->count * sizeof(float));
    float* devAvgGradRotations;
    hipMalloc(&devAvgGradRotations, model->count * 4 * sizeof(float));
    hipMemset(devAvgGradRotations, 0, model->count * 4 * sizeof(float));

    float* devLossPixels;
    hipMalloc(&devLossPixels, RENDER_RESOLUTION_X * RENDER_RESOLUTION_Y * 3 * sizeof(float));

    float* devGradLocations;
    hipMalloc(&devGradLocations, model->count * 3 * sizeof(float));
    float* devGradShs;
    hipMalloc(&devGradShs, model->count * 3 * model->shCoeffs * sizeof(float));
    float* devGradScales;
    hipMalloc(&devGradScales, model->count * 3 * sizeof(float));
    float* devGradOpacities;
    hipMalloc(&devGradOpacities, model->count * sizeof(float));
    float* devGradRotations;
    hipMalloc(&devGradRotations, model->count * 4 * sizeof(float));

    float* devGradMean2D;
    hipMalloc(&devGradMean2D, model->count * 3 * sizeof(float));
    float* devGradConic;
    hipMalloc(&devGradConic, model->count * 4 * sizeof(float));
    float* devGradColor;
    hipMalloc(&devGradColor, model->count * 3 * sizeof(float));
    float* devGradCov3D;
    hipMalloc(&devGradCov3D, model->count * 6 * sizeof(float));

    for (int i = 0; i < truthFrameBuffers.size(); i++) {
        uint32_t* truthFrameBuffer = truthFrameBuffers[i];
        Camera camera = truthCameras[i];

        std::vector<float> background = {0.0f, 0.0f, 0.0f};
        hipMemcpy(devBackground, background.data(), 3 * sizeof(float), hipMemcpyHostToDevice);

        glm::mat4 matView = -glm::lookAt(TruthCameras::toGlmVec(camera.location),
                                         TruthCameras::toGlmVec(camera.target), {0.0f, 1.0f, 0.0f});
        hipMemcpy(devMatView, glm::value_ptr(matView), 16 * sizeof(float), hipMemcpyHostToDevice);

        glm::mat4 matProjView = glm::perspective(glm::radians(camera.degFovY),
                                                 (float)RENDER_RESOLUTION_X / (float)RENDER_RESOLUTION_Y, 0.1f, 100.0f) * matView;
        hipMemcpy(devMatProjView, glm::value_ptr(matProjView), 16 * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(devCameraLocation, &camera.location[0], 3 * sizeof(float), hipMemcpyHostToDevice);

        char* geomBuffer;
        char* binningBuffer;
        char* imgBuffer;

        int countRendered = CudaRasterizer::Rasterizer::forward(
                [&](size_t N) { hipMalloc(&geomBuffer, N); return geomBuffer; },
                [&](size_t N) { hipMalloc(&binningBuffer, N); return binningBuffer; },
                [&](size_t N) { hipMalloc(&imgBuffer, N); return imgBuffer; },
                model->count,
                model->shDegree,
                model->shCoeffs,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devOpacities,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                false,
                devRasterized,
                nullptr,
                true);

        lossKernel<<<256, 256>>>(truthFrameBuffer, devRasterized, devLossPixels, 256 * 256, RENDER_RESOLUTION_X, RENDER_RESOLUTION_Y);

        hipMemset(devGradLocations, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradShs, 0, model->count * 3 * model->shCoeffs * sizeof(float));
        hipMemset(devGradScales, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradOpacities, 0, model->count * sizeof(float));
        hipMemset(devGradRotations, 0, model->count * 4 * sizeof(float));

        hipMemset(devGradMean2D, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradConic, 0, model->count * 4 * sizeof(float));
        hipMemset(devGradColor, 0, model->count * 3 * sizeof(float));
        hipMemset(devGradCov3D, 0, model->count * 6 * sizeof(float));

        CudaRasterizer::Rasterizer::backward(
                model->count,
                model->shDegree,
                model->shCoeffs,
                countRendered,
                devBackground,
                RENDER_RESOLUTION_X,
                RENDER_RESOLUTION_Y,
                model->devLocations,
                model->devShs,
                nullptr,
                model->devScales,
                1.0f,
                model->devRotations,
                nullptr,
                devMatView,
                devMatProjView,
                devCameraLocation,
                tan(glm::radians(camera.degFovX) * 0.5f),
                tan(glm::radians(camera.degFovY) * 0.5f),
                nullptr, // TODO pass in from forward rasterizer to increase speed
                geomBuffer,
                binningBuffer,
                imgBuffer,
                devLossPixels,
                devGradMean2D,
                devGradConic,
                devGradOpacities,
                devGradColor,
                devGradLocations,
                devGradCov3D,
                devGradShs,
                devGradScales,
                devGradRotations,
                true);

        gradientSumKernel<<<256, 256>>>(devAvgGradLocations, devAvgGradShs, devAvgGradScales, devAvgGradOpacities, devAvgGradRotations,
                                        devGradLocations, devGradShs, devGradScales, devGradOpacities, devGradRotations,
                                        (float)truthFrameBuffers.size(), model->shCoeffs, 256 * 256, model->count);

        hipFree(geomBuffer);
        hipFree(binningBuffer);
        hipFree(imgBuffer);
    }

    hipFree(devLossPixels);

    hipFree(devGradLocations);
    hipFree(devGradShs);
    hipFree(devGradScales);
    hipFree(devGradOpacities);
    hipFree(devGradRotations);

    hipFree(devGradMean2D);
    hipFree(devGradConic);
    hipFree(devGradColor);
    hipFree(devGradCov3D);

    float* avgGradLocations = new float[model->count * 3];
    hipMemcpy(avgGradLocations, devAvgGradLocations, model->count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    float* avgGradShs = new float[model->count * 3 * model->shCoeffs];
    hipMemcpy(avgGradShs, devAvgGradShs, model->count * 3 * model->shCoeffs * sizeof(float), hipMemcpyDeviceToHost);
    float* avgGradScales = new float[model->count * 3];
    hipMemcpy(avgGradScales, devAvgGradScales, model->count * 3 * sizeof(float), hipMemcpyDeviceToHost);
    float* avgGradOpacities = new float[model->count];
    hipMemcpy(avgGradOpacities, devAvgGradOpacities, model->count * sizeof(float), hipMemcpyDeviceToHost);
    float* avgGradRotations = new float[model->count * 4];
    hipMemcpy(avgGradRotations, devAvgGradRotations, model->count * 4 * sizeof(float), hipMemcpyDeviceToHost);

    static const float learningRate = 0.02f;

    for (int i = 0; i < model->count; i++) {
        for(int f = 0; f < 3; f++) {
            model->locations[i * 3 + f] += avgGradLocations[i * 3 + f] * learningRate;
        }
        for(int f  = 0; f < model->shCoeffs * 3; f++) {
            model->shs[i * 3 * model->shCoeffs + f] += avgGradShs[i * 3 * model->shCoeffs + f] * learningRate;
        }
        for(int f = 0; f < 3; f++) {
            model->scales[i * 3 + f] += avgGradScales[i * 3 + f] * learningRate;
            model->scales[i * 3 + f] = std::max(0.0f, model->scales[i * 3 + f]);
        }
        model->opacities[i] = std::max(1.0f, std::min(0.0f, model->opacities[i] + avgGradOpacities[i] * learningRate));
        for(int f = 0; f < 4; f++) {
            model->rotations[i * 4 + f] += avgGradRotations[i * 4 + f] * learningRate;
        }
    }

    hipFree(devAvgGradLocations);
    hipFree(devAvgGradShs);
    hipFree(devAvgGradScales);
    hipFree(devAvgGradOpacities);
    hipFree(devAvgGradRotations);

    delete[] avgGradLocations;
    delete[] avgGradShs;
    delete[] avgGradScales;
    delete[] avgGradOpacities;
    delete[] avgGradRotations;

}
